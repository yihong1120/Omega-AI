
#include <hip/hip_runtime.h>
#define BLOCK 1024 
#define ETA 10e-8 

extern "C"
__global__ void adam(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    //float tmp = diffW[i] / batch;
    float tmp = diffW[i];
    mw[i] = beta1 * mw[i] + (1 - beta1) * tmp;
	vw[i] = beta2 * vw[i] + (1 - beta2) * tmp * tmp;
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate * mhat / (sqrt(vhat) + ETA);
}

extern "C"
__global__ void adamw(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, float weight_decay, int n, int batch, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    //float tmp = diffW[i] / batch;
    float tmp = diffW[i];
    float theta = weight[i] - learnRate * weight_decay * weight[i];
    mw[i] = beta1 * mw[i] + (1 - beta1) * tmp;
	vw[i] = beta2 * vw[i] + (1 - beta2) * tmp * tmp;
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = theta - learnRate * mhat / (sqrt(vhat) + ETA);
}

extern "C"
__global__ void adam_bn(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, int n, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    mw[i] = beta1 * mw[i] + (1 - beta1) * diffW[i];
	vw[i] = beta2 * vw[i] + (1 - beta2) * diffW[i] * diffW[i];
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate * mhat / (sqrt(vhat) + ETA);
}

extern "C"
__global__ void adamw_bn(float *diffW, float *weight,float *mw,float *vw,float beta1,float beta2,float learnRate, float weight_decay, int n, int t)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    //float theta = weight[i] - learnRate * weight_decay * weight[i];
    mw[i] = beta1 * mw[i] + (1 - beta1) * diffW[i];
	vw[i] = beta2 * vw[i] + (1 - beta2) * diffW[i] * diffW[i];
	float mhat = mw[i] / (1 - powf(beta1, t));
	float vhat = vw[i] / (1 - powf(beta2, t));
	weight[i] = weight[i] - learnRate * mhat / (sqrt(vhat) + ETA);
}


extern "C"
__global__ void sgd(float *diffW, float *v,float *weight,float momentum,float weight_decay,float learnRate, int n, int batch)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    //float tmp = (diffW[i] / batch) + weight[i] * weight_decay;
    float tmp = diffW[i] + weight[i] * weight_decay;
    v[i] = v[i] * momentum + tmp;
	weight[i] = weight[i] - learnRate * v[i];
}

extern "C"
__global__ void sgd_bn(float *diffW, float *v,float *weight,float momentum,float weight_decay,float learnRate, int n, int batch)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    //float tmp = (diffW[i] / batch) + weight[i] * weight_decay;
    float tmp = diffW[i] + weight[i] * weight_decay;
    v[i] = v[i] * momentum + tmp;
	weight[i] = weight[i] - learnRate * v[i];
}